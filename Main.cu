
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>

#include <cudnn.h>
#include <hipblas.h>

int main() {
  cudnnHandle_t cudnn;
  cudnnCreate(&cudnn);
  
  cudnnTensorDescriptor_t inputDesc;
  cudnnTensorDescriptor_t outputDesc;
  cudnnFilterDescriptor_t kernelDesc;
  
  cudnnCreateTensorDescriptor(&inputDesc);
  cudnnCreateTensorDescriptor(&outputDesc);
  cudnnCreateFilterDescriptor(&kernelDesc);
  
  cudnnSetTensor4dDescriptor(inputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, 1, 4, 4);
  cudnnSetTensor4dDescriptor(outputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, 1, 2, 2, 2);
  cudnnSetFilter4dDescriptor(kernelDesc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, 2, 1, 3, 3);
  
  cudnnConvolutionDescriptor_t convDesc;
  cudnnCreateConvolutionDescriptor(&convDesc);
  cudnnSetConvolution2dDescriptor(convDesc, 0, 0, 1, 1, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);
  
  int maxConvAlgos = 1;
  cudnnConvolutionFwdAlgoPerf_t convFwdAlgos[maxConvAlgos];
  cudnnConvolutionBwdDataAlgoPerf_t convBwdDataAlgos[maxConvAlgos];
  cudnnConvolutionBwdFilterAlgoPerf_t convBwdFilterAlgos[maxConvAlgos];
  cudnnFindConvolutionForwardAlgorithm(cudnn, inputDesc, kernelDesc, convDesc, outputDesc, maxConvAlgos, &maxConvAlgos, convFwdAlgos);
  cudnnFindConvolutionBackwardDataAlgorithm(cudnn, kernelDesc, outputDesc, convDesc, inputDesc, maxConvAlgos, &maxConvAlgos, convBwdDataAlgos);
  cudnnFindConvolutionBackwardFilterAlgorithm(cudnn, inputDesc, outputDesc, convDesc, kernelDesc, maxConvAlgos, &maxConvAlgos, convBwdFilterAlgos);
  cudnnConvolutionFwdAlgo_t convFwdAlgo = convFwdAlgos[0].algo;
  cudnnConvolutionBwdDataAlgo_t convBwdDataAlgo = convBwdDataAlgos[0].algo;
  cudnnConvolutionBwdFilterAlgo_t convBwdFilterAlgo = convBwdFilterAlgos[0].algo;
  printf("ConvFwdAlgo: %d\n\n", convFwdAlgo);
  printf("ConvBwdDataAlgo: %d\n\n", convBwdDataAlgo);
  printf("ConvBwdFilterAlgo: %d\n\n", convBwdFilterAlgo);
  
  
  float* dInputTensor;
  float* dOutputTensor;
  float* dKernelTensor;
  
  float* dInputGradTensor;
  float* dOutputGradTensor;
  float* dKernelGradTensor;
  
  hipMalloc(&dInputTensor, 4 * 4 * 1 * 1 * sizeof(float));
  hipMalloc(&dOutputTensor, 2 * 2 * 2 * 1 * sizeof(float));
  hipMalloc(&dKernelTensor, 3 * 3 * 1 * 2 * sizeof(float));
  
  hipMalloc(&dInputGradTensor, 4 * 4 * 1 * 1 * sizeof(float));
  hipMalloc(&dOutputGradTensor, 2 * 2 * 2 * 1 * sizeof(float));
  hipMalloc(&dKernelGradTensor, 3 * 3 * 1 * 2 * sizeof(float));
  
  float hInputTensor[4 * 4 * 1 * 1] = {
    1, 2, 3, 4,
    5, 6, 7, 8,
    9,10,11,12,
    13,14,15,16
  };
  
  float hKernelTensor[3 * 3 * 1 * 2] = {
    1, 2, 3,
    4, 5, 6,
    7, 8, 9,
    -1, -2, -3,
    -4, -5, -6,
    -7, -8, -9
  };
  
  float hOutputTensor[2 * 2 * 2 * 1];
  
  float hOutputGradTensor[2 * 2 * 2 * 1] = {
    -1, -2,
    -3, -4,
    1, 2,
    3, 4
  };
    
  float hInputGradTensor[4 * 4 * 1 * 1];
  float hKernelGradTensor[3 * 3 * 1 * 2];
  
  hipMemcpy(dInputTensor, hInputTensor, 4 * 4 * 1 * 1 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dKernelTensor, hKernelTensor, 3 * 3 * 1 * 2 * sizeof(float), hipMemcpyHostToDevice);
  
  const float alpha = 1.0f;
  const float beta = 0.0f;
  cudnnConvolutionForward(
    cudnn,
    &alpha,
    inputDesc, dInputTensor,
    kernelDesc, dKernelTensor,
    convDesc, convFwdAlgo,
    0, 0,
    &beta,
    outputDesc, dOutputTensor);
  
  hipMemcpy(hOutputTensor, dOutputTensor, 2 * 2 * 2 * 1 * sizeof(float), hipMemcpyDeviceToHost);
  
  printf("hOutputTensor:\n");
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < 2; ++j) {
      for (int k = 0; k < 2; ++k) {
        for (int l = 0; l < 2; ++l) {
          printf("%f ", hOutputTensor[i * 2 * 2 * 2 + j * 2 * 2 + k * 2 + l]);
        }
        printf("\n");
      }
      printf("\n");
    }
    printf("\n");
  }
  
  hipMemcpy(dOutputGradTensor, hOutputGradTensor, 2 * 2 * 2 * 1 * sizeof(float), hipMemcpyHostToDevice);
  
  cudnnConvolutionBackwardData(
    cudnn,
    &alpha,
    kernelDesc, dKernelTensor,
    outputDesc, dOutputGradTensor,
    convDesc, convBwdDataAlgo,
    0, 0,
    &beta,
    inputDesc, dInputGradTensor);
    
    cudnnConvolutionBackwardFilter(
    cudnn,
    &alpha,
    inputDesc, dInputTensor,
    outputDesc, dOutputGradTensor,
    convDesc, convBwdFilterAlgo,
    0, 0,
    &beta,
    kernelDesc, dKernelGradTensor);
  
  hipMemcpy(hInputGradTensor, dInputGradTensor, 4 * 4 * 1 * 1 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(hKernelGradTensor, dKernelGradTensor, 3 * 3 * 1 * 2 * sizeof(float), hipMemcpyDeviceToHost);
  
  printf("hInputGradTensor:\n");
  for (int i = 0; i < 1; ++i) {
    for (int j = 0; j < 1; ++j) {
      for (int k = 0; k < 4; ++k) {
        for (int l = 0; l < 4; ++l) {
          printf("%f ", hInputGradTensor[i * 4 * 4 * 1 + j * 4 * 4 + k * 4 + l]);
        }
        printf("\n");
      }
      printf("\n");
    }
    printf("\n");
  }
  
  printf("hKernelGradTensor:\n");
  for (int i = 0; i < 2; ++i) {
    for (int j = 0; j < 1; ++j) {
      for (int k = 0; k < 3; ++k) {
        for (int l = 0; l < 3; ++l) {
          printf("%f ", hKernelGradTensor[i * 3 * 3 * 1 + j * 3 * 3 + k * 3 + l]);
        }
        printf("\n");
      }
      printf("\n");
    }
    printf("\n");
  }
  
  return 0;
}