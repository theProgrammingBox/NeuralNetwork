#include "hip/hip_runtime.h"
#include "Header.cuh"

/*
LESSONS LEARNED:
- shallower network needs more epochs given simple task as deeper networks can quickly compound changes(hypoth)
- adding more depth to network can help with more complex tasks or create more complex dynamic in my case
- massive and miniscule gradients are possible with my algo due to the value network modeling the "reward" of the policy network
- in my algo, the value nn forgets senarios it has not seen in a while, will need a way to solve this
-- maybe alot more diversity so it doesn't really need to remember as long as it can keep up with modern senarios
- currently policy has the behavior of just exploding to out of bounds actions, has to do with value nn not reflecting the true reward gradient initially(hypoth)
-- tried training value nn first 1024 epoch, then policy nn, but it shows 0 gradient, see if adam can help. can be the extreme gradients of the learned value nn
-- tried setting the score for out of bounds actions to 0, but it still kind of doesn't work
--- the problem is that the slope at the edges are not steering towards the center all the time because it is basically learning a step function that is not exactly flat.
*/

/*
TASKS:
- visualize data on different application
- simulate rps to see if it can handle basic changing strategies
-- maybe see oscillating strategies. see if stagnates due to the nature of nn or value function failure
- add cuda memset for 0s called zero
*/

struct network {
  float meanCorrection;
  float varianceCorrection;
  uint32_t batchSize;
  uint32_t layers;
  uint32_t* parameters;
  float** outputs;
  float** weights;
  float** outputGradients;
  float** weightGradients;
  float** weightGradientsMean;
  float** weightGradientsVariance;
};

void initializeNetwork(network* net, const uint32_t batchSize, const uint32_t layers, uint32_t* parameters, uint32_t *seed1, uint32_t *seed2, bool debug = false) {
  net->batchSize = batchSize;
  net->layers = layers;
  net->parameters = parameters;
  
  net->meanCorrection = 1.0f;
  net->varianceCorrection = 1.0f;
  
  net->outputs = (float**)malloc((net->layers + 2) * sizeof(float*));
  net->outputGradients = (float**)malloc((net->layers + 2) * sizeof(float*));
  
  if (debug) printf("Initialize network:\n");
  for (uint32_t i = 0; i < net->layers + 2; i++) {
    checkCudaStatus(hipMalloc(&net->outputs[i], net->parameters[i] * net->batchSize * sizeof(float)));
    if (debug) printDTensor(net->outputs[i], net->parameters[i], net->batchSize, "output");
    
    checkCudaStatus(hipMalloc(&net->outputGradients[i], net->parameters[i] * net->batchSize * sizeof(float)));
    if (debug) printDTensor(net->outputGradients[i], net->parameters[i], net->batchSize, "output gradient");
  }
  
  net->weights = (float**)malloc((net->layers + 1) * sizeof(float*));
  net->weightGradients = (float**)malloc((net->layers + 1) * sizeof(float*));
  net->weightGradientsMean = (float**)malloc((net->layers + 1) * sizeof(float*));
  net->weightGradientsVariance = (float**)malloc((net->layers + 1) * sizeof(float*));
  
  for (uint32_t i = 0; i < net->layers + 1; i++) {
    checkCudaStatus(hipMalloc(&net->weights[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    fillDTensor(net->weights[i], net->parameters[i + 1] * net->parameters[i], seed1, seed2);
    if (debug) printDTensor(net->weights[i], net->parameters[i + 1], net->parameters[i], "weight");
    
    checkCudaStatus(hipMalloc(&net->weightGradients[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    if (debug) printDTensor(net->weightGradients[i], net->parameters[i + 1], net->parameters[i], "weight gradient");
    
    checkCudaStatus(hipMalloc(&net->weightGradientsMean[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    customFillDTensorConstant(net->weightGradientsMean[i], net->parameters[i + 1] * net->parameters[i], 0.0f);
    if (debug) printDTensor(net->weightGradientsMean[i], net->parameters[i + 1], net->parameters[i], "weight gradient mean");
    
    checkCudaStatus(hipMalloc(&net->weightGradientsVariance[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    customFillDTensorConstant(net->weightGradientsVariance[i], net->parameters[i + 1] * net->parameters[i], 0.0f);
    if (debug) printDTensor(net->weightGradientsVariance[i], net->parameters[i + 1], net->parameters[i], "weight gradient variance");
  }
  if (debug) printf("\n");
}

void setRandomInput(network* net, uint32_t *seed1, uint32_t *seed2) {
    fillDTensor(net->outputs[0], net->parameters[0] * net->batchSize, seed1, seed2);
}

void setCustomRandomInput(network* net, uint32_t *seed1, uint32_t *seed2) {
  customFillDTensor(net->outputs[0], net->parameters[0] * net->batchSize, seed1, seed2);
}

void setInput(network* net, float* inputs, bool host = true) {
  checkCudaStatus(hipMemcpy(net->outputs[0], inputs, net->parameters[0] * net->batchSize * sizeof(float), host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}

void forwardPropagate(hipblasHandle_t *cublasHandle, network* net, bool debug = false) {
  const float zero = 0.0f;
  if (debug) printf("Forward propagation:\n");
  if (debug) printDTensor(net->outputs[0], net->parameters[0], net->batchSize, "input");
  
  for (uint32_t i = 0; i < net->layers; i++) {
    // float alpha = 2.0f / sqrtf(net->parameters[i]);
    float alpha = 1.0f;
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
      net->parameters[i + 1], net->batchSize, net->parameters[i],
      &alpha,
      net->weights[i], net->parameters[i + 1],
      net->outputs[i], net->parameters[i],
      &zero,
      net->outputs[i + 1], net->parameters[i + 1]));
    if (debug) printDTensor(net->outputs[i + 1], net->parameters[i + 1], net->batchSize, "sum");
    
    reluForward(net->outputs[i + 1], net->parameters[i + 1] * net->batchSize);
    if (debug) printDTensor(net->outputs[i + 1], net->parameters[i + 1], net->batchSize, "relu");
  }
  
  // float alpha = 2.0f / sqrtf(net->parameters[net->layers]);
  float alpha = 1.0f;
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    net->parameters[net->layers + 1], net->batchSize, net->parameters[net->layers],
    &alpha,
    net->weights[net->layers], net->parameters[net->layers + 1],
    net->outputs[net->layers], net->parameters[net->layers],
    &zero,
    net->outputs[net->layers + 1], net->parameters[net->layers + 1]));
  if (debug) printDTensor(net->outputs[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output");
  
  if (debug) printf("\n");
}

void setOutputTarget(hipblasHandle_t *cublasHandle, network* net, float* target, bool debug = false) {
  checkCudaStatus(hipMemcpy(net->outputGradients[net->layers + 1], target, net->parameters[net->layers + 1] * net->batchSize * sizeof(float), hipMemcpyHostToDevice));
  const float negativeOne = -1.0f;
  checkCublasStatus(hipblasSaxpy(
    *cublasHandle,
    net->parameters[net->layers + 1] * net->batchSize,
    &negativeOne,
    net->outputs[net->layers + 1], 1,
    net->outputGradients[net->layers + 1], 1));
  if (debug) printDTensor(net->outputGradients[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output gradient");
}

void setOutputGradients(network* net, float* gradients, bool host = true) {
  checkCudaStatus(hipMemcpy(net->outputGradients[net->layers + 1], gradients, net->parameters[net->layers + 1] * net->batchSize * sizeof(float), host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}

void setOutputGradientsToConstant(network* net, float constant) {
  customFillDTensorConstant(net->outputGradients[net->layers + 1], net->parameters[net->layers + 1] * net->batchSize, constant);
}

void backPropagate(hipblasHandle_t *cublasHandle, network* net, bool errorPrint = false, bool debug = false) {
  if (errorPrint) {
    float error = 0.0f;
    checkCublasStatus(hipblasSasum(
      *cublasHandle,
      net->parameters[net->layers + 1] * net->batchSize,
      net->outputGradients[net->layers + 1], 1,
      &error));
    printf("Error: %f\n", error / net->batchSize);
  }
  
  const float zero = 0.0f;
  if (debug) printf("Back propagation:\n");
  
  if (debug) printDTensor(net->outputGradients[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output gradient");
  
  // float alpha = 2.0f / sqrtf(net->batchSize);
  float alpha = 1.0f;
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
    net->parameters[net->layers + 1], net->parameters[net->layers], net->batchSize,
    &alpha,
    net->outputGradients[net->layers + 1], net->parameters[net->layers + 1],
    net->outputs[net->layers], net->parameters[net->layers],
    &zero,
    net->weightGradients[net->layers], net->parameters[net->layers + 1]));
  if (debug) printDTensor(net->weightGradients[net->layers], net->parameters[net->layers + 1], net->parameters[net->layers], "weight gradient");
  
  // float beta = 2.0f / sqrtf(net->parameters[net->layers + 1]);
  float beta = 1.0f;
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
    net->parameters[net->layers], net->batchSize, net->parameters[net->layers + 1],
    &beta,
    net->weights[net->layers], net->parameters[net->layers + 1],
    net->outputGradients[net->layers + 1], net->parameters[net->layers + 1],
    &zero,
    net->outputGradients[net->layers], net->parameters[net->layers]));
  if (debug) printDTensor(net->outputGradients[net->layers], net->parameters[net->layers], net->batchSize, "output gradient");
  
  for (uint32_t i = net->layers; i > 0; i--) {
    reluBackward(net->outputs[i], net->outputGradients[i], net->parameters[i] * net->batchSize);
    if (debug) printDTensor(net->outputGradients[i], net->parameters[i], net->batchSize, "relu gradient");
    
    // alpha = 2.0f / sqrtf(net->batchSize);
    alpha = 1.0f;
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      net->parameters[i], net->parameters[i - 1], net->batchSize,
      &alpha,
      net->outputGradients[i], net->parameters[i],
      net->outputs[i - 1], net->parameters[i - 1],
      &zero,
      net->weightGradients[i - 1], net->parameters[i]));
    if (debug) printDTensor(net->weightGradients[i - 1], net->parameters[i], net->parameters[i - 1], "weight gradient");
    
    // beta = 2.0f / sqrtf(net->parameters[i]);
    beta = 1.0f;
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      net->parameters[i - 1], net->batchSize, net->parameters[i],
      &beta,
      net->weights[i - 1], net->parameters[i],
      net->outputGradients[i], net->parameters[i],
      &zero,
      net->outputGradients[i - 1], net->parameters[i - 1]));
    if (debug) printDTensor(net->outputGradients[i - 1], net->parameters[i - 1], net->batchSize, "output gradient");
  }
  if (debug) printf("\n");
}

void updateWeights(hipblasHandle_t *cublasHandle, network* net, float betaMean, float betaVar, float learningRate, float weightDecay, bool debug = false) {
  if (debug) printf("Update weights:\n");
  net->meanCorrection *= betaMean;
  net->varianceCorrection *= betaVar;
  for (uint32_t i = 0; i < net->layers + 1; i++) {
    integratedAdamUpdate(net->weights[i], net->weightGradients[i], net->weightGradientsMean[i], net->weightGradientsVariance[i], betaMean, betaVar, net->meanCorrection, net->varianceCorrection, learningRate, weightDecay, net->parameters[i + 1] * net->parameters[i]);
    if (debug) printDTensor(net->weights[i], net->parameters[i + 1], net->parameters[i], "weight");
  }
  if (debug) printf("\n");
}

void freeNetwork(network* net, bool debug = false) {
  if (debug) printf("Free network\n");
  for (uint32_t i = 0; i < net->layers; i++)
    checkCudaStatus(hipFree(net->weights[i]));
  
  free(net->weights);
  
  for (uint32_t i = 0; i < net->layers + 1; i++)
    checkCudaStatus(hipFree(net->outputs[i]));
  
  free(net->outputs);
}

int main() {
  uint32_t seed1, seed2;
  initializeSeeds(&seed1, &seed2);
  
  hipblasHandle_t handle;
  checkCublasStatus(hipblasCreate(&handle));
  
  const uint32_t epochs = 4096 * 16;
  const uint32_t batchSize = 4096;
  const float meanBeta = 0.9f;
  const float varianceBeta = 0.999f;
  const float weightDecay = 0.2f;
  const float policyLearningRate = 0.1f / sqrtf(batchSize);
  const float valueLearningRate = 0.1f / sqrtf(batchSize);
  
  network policy;
  uint32_t policyParameters[] = {2, 8, 8, 1};
  const uint32_t policyLayers = sizeof(policyParameters) / sizeof(uint32_t) - 2;
  initializeNetwork(&policy, batchSize, policyLayers, policyParameters, &seed1, &seed2);
  
  network value;
  uint32_t valueParameters[] = {2, 8, 8, 1};
  const uint32_t valueLayers = sizeof(valueParameters) / sizeof(uint32_t) - 2;
  initializeNetwork(&value, batchSize, valueLayers, valueParameters, &seed1, &seed2);
  
  float policyOutput[policyParameters[policyLayers + 1] * batchSize];
  float valueTarget[policyParameters[policyLayers + 1] * batchSize];
  uint32_t actions[batchSize];
  const int outcomes[9] = {
    0, -1,  1,
    1,  0, -1,
    -1,  1,  0
  };
  
  float valueInput[valueParameters[0] * batchSize];
  for (uint32_t epoch = 0; epoch < epochs; epoch++) {
    setCustomRandomInput(&policy, &seed1, &seed2);
    forwardPropagate(&handle, &policy);
    
    checkCudaStatus(hipMemcpy(policyOutput, policy.outputs[policyLayers + 1], policyParameters[policyLayers + 1] * batchSize * sizeof(float), hipMemcpyDeviceToHost));
    for (uint32_t batch = 0; batch < batchSize; batch++) {
      // float in = generateRandomFloat(&seed1, &seed2) * 0.2f;
      float in = policyOutput[batch];// + generateRandomFloat(&seed1, &seed2);
      valueInput[batch * 2] = in;
      valueInput[batch * 2 + 1] = 1.0f;
      valueTarget[batch] = -((0.1f - in) * (0.1f - in));
    }
    
    
    setInput(&value, valueInput);
    forwardPropagate(&handle, &value);
    // if (epoch < epochs / 2) {
      setOutputTarget(&handle, &value, valueTarget);
      backPropagate(&handle, &value, epoch % 1024 == 0);
      updateWeights(&handle, &value, meanBeta, varianceBeta, valueLearningRate, weightDecay);
    // }
    
    // if (epoch > epochs / 2) {
      setOutputGradientsToConstant(&value, 1.0f);
      backPropagate(&handle, &value);
      setOutputGradients(&policy, value.outputGradients[0], false);
      backPropagate(&handle, &policy);
      if (epoch == epochs - 1) {
        updateWeights(&handle, &policy, meanBeta, varianceBeta, policyLearningRate, weightDecay, true);
      } else {
        updateWeights(&handle, &policy, meanBeta, varianceBeta, policyLearningRate, weightDecay);
      }
    // }
  }
  printf("\n");
  
  float policyInput[policyParameters[0] * batchSize];
  for (uint32_t point = 0; point < 21; point++) {
    policyInput[0] = 1;
    policyInput[1] = ((float)point- 10) / 10.0f;
    policyInput[2] = 1.0f;
    setInput(&policy, policyInput);
    forwardPropagate(&handle, &policy);
    
    float policyOutput[policyParameters[policyLayers + 1] * batchSize];
    checkCudaStatus(hipMemcpy(policyOutput, policy.outputs[policyLayers + 1], policyParameters[policyLayers + 1] * batchSize * sizeof(float), hipMemcpyDeviceToHost));
    
    float input;
    float valuet;
    
    float in = policyOutput[0];
    valuet = -((0.1f-in) * (0.1f-in));
    
    valueInput[0] = in;
    valueInput[1] = 1.0f;
    setInput(&value, valueInput);
    forwardPropagate(&handle, &value);
    setOutputGradientsToConstant(&value, 1.0f);
    backPropagate(&handle, &value);
    
    float valueGradient[valueParameters[0] * batchSize];
    checkCudaStatus(hipMemcpy(valueGradient, value.outputGradients[0], valueParameters[0] * batchSize * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("In: %f, Out: %f, Score: %f, Grad: %f\n", policyInput[1], policyOutput[0], valuet, valueGradient[0]);
  }
  printf("\n");
  
  for (uint32_t point = 0; point < 21; point++) {
    valueInput[0] = ((float)point- 10) / (10.0f / 0.4f);
    setInput(&value, valueInput);
    forwardPropagate(&handle, &value);
    
    float valueOutput[valueParameters[valueLayers + 1] * batchSize];
    checkCudaStatus(hipMemcpy(valueOutput, value.outputs[valueLayers + 1], valueParameters[valueLayers + 1] * batchSize * sizeof(float), hipMemcpyDeviceToHost));
    
    float valueGradient[valueParameters[valueLayers + 1] * batchSize];
    
    setOutputGradientsToConstant(&value, 1.0f);
    backPropagate(&handle, &value);
    
    float valueInputGradient[valueParameters[valueLayers + 1] * batchSize];
    checkCudaStatus(hipMemcpy(valueInputGradient, value.outputGradients[0], valueParameters[valueLayers + 1] * batchSize * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("In: %f, Out: %f, expected: %f, Grad: %f\n", valueInput[0], valueOutput[0], -((0.1f - valueInput[0]) * (0.1f - valueInput[0])), valueInputGradient[0]);
  }
  
  freeNetwork(&policy);
  freeNetwork(&value);
  
  
  checkCublasStatus(hipblasDestroy(handle));

  return 0;
}