#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void fillOne(int size, float *data) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= size) return;
	data[i] = 1;
}

__global__ void prefixSum(int size, float *data, int itr) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int threads = blockDim.x;
	int temp;
	int exp = 1;
	for (int j = 0; j < itr; j++) {
		for (int k = 0; k < size; k++) {
			temp = threads * k + i * exp * 2 - 1;
			if (temp < size) {
				data[temp] = data[temp] + data[temp - exp];
			}
		}
		__syncthreads();
		exp *= 2;
	}
	
	for (int j = 0; j < itr; j++) {
		exp >>= 1;
		temp = i * exp * 2 - 1 + exp;
		if (temp < size) {
			data[temp] = data[temp] + data[temp - exp];
		}
		__syncthreads();
	}
}

void printDataDev(float *dataDev, int size) {
	float *data = (float *)malloc(size * sizeof(float));
	hipMemcpy(data, dataDev, size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < size; i++) {
		printf("%.0f, ", data[i]);
	}
	printf("\n");
	free(data);
}

int main(int argc, char *argv[])
{
	const int size = 100;
	float *dataDev;
	
	hipMalloc(&dataDev, size * sizeof(float));
	
	fillOne<<<1, 16>>>(size, dataDev);
	
	prefixSum<<<1, 16>>>(size, dataDev, floor(log2(size)));
	printDataDev(dataDev, size);
	printf("floor(log2(size)) = %.0f\n", floor(log2(size)));
	
	hipFree(dataDev);
	
	return 0;
}