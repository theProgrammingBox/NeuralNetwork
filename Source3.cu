
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

void initializeSeeds(uint32_t *seed1, uint32_t *seed2) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    *seed1 = tv.tv_sec;
    *seed2 = tv.tv_usec;
    for (uint8_t i = 8; i--;) {
        *seed2 *= 0xbf324c81;
        *seed1 ^= *seed2;
        *seed1 ^= *seed1 >> 13;
        *seed1 *= 0x9c7493ad;
        *seed2 ^= *seed1;
        *seed2 ^= *seed2 >> 17;
    }
}

__global__ void _fillDTensor(float *dTensor, uint32_t size, uint32_t seed1, uint32_t seed2) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
	seed1 ^= idx;
    seed1 *= 0x4ba1bb47;
    seed1 ^= seed1 >> 17;
    seed1 ^= seed2;
    seed1 *= 0xb7ebcb79;
    seed1 ^= seed1 >> 13;
    dTensor[idx] = (int32_t)seed1 * 0.0000000004656612875245797f;
}

void fillDTensor(float *dTensor, uint32_t size, uint32_t *seed1, uint32_t *seed2) {
    *seed2 *= 0xbf324c81;
    *seed1 ^= *seed2;
    *seed1 ^= *seed1 >> 13;
    *seed1 *= 0x9c7493ad;
    *seed2 ^= *seed1;
    *seed2 ^= *seed2 >> 17;
    _fillDTensor<<<(size >> 10) + (size & 0x3ff), 0x400>>>(dTensor, size, *seed1, *seed2);
}

void printDTensor(float *dTensor, uint32_t width, uint32_t height, const char *label) {
    float *tensor = (float *)malloc(width * height * sizeof(float));
    checkCudaStatus(hipMemcpy(tensor, dTensor, width * height * sizeof(float), hipMemcpyDeviceToHost));
    printf("%s:\n", label);
    for (uint32_t i = 0; i < height; i++) {
        for (uint32_t j = 0; j < width; j++) {
            printf("%f ", tensor[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
    free(tensor);
}

void mixSeed(uint32_t *seed1, uint32_t *seed2) {
    *seed2 *= 0xbf324c81;
    *seed1 ^= *seed2;
    *seed1 ^= *seed1 >> 17;
    *seed1 *= 0x9c7493ad;
    *seed2 ^= *seed1;
    *seed2 ^= *seed2 >> 13;
}

__global__ void _addDTensors(float alpha, float *dTensor1, float *dTensor2, uint32_t size) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
    dTensor1[idx] += alpha * dTensor2[idx];
}

void addDTensors(float alpha, float *dTensor1, float *dTensor2, uint32_t size) {
    _addDTensors<<<(size >> 10) + (size & 0x3ff), 0x400>>>(alpha, dTensor1, dTensor2, size);
}

int main() {
    const uint16_t networkBatches = 1024;
    const uint8_t networkLayers = 3;
    uint8_t networkParameters[networkLayers + 1] = {3, 4, 4, 1};
    float* weightTensors[networkLayers], * outputTensors[networkLayers + 1];
    float* outputGradTensors[networkLayers + 1], * weightGradTensors[networkLayers];
    uint32_t seed1, seed2;
    hipblasLtHandle_t ltHandle;
    hipblasLtMatmulDesc_t OutputOpDesc, InputGradOpDesc, WeightGradOpDesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatrixLayout_t weightLayouts[networkLayers], outputLayouts[networkLayers + 1];
    hipblasLtMatmulAlgo_t outputAlgos[networkLayers], inputGradAlgos[networkLayers], weightGradAlgos[networkLayers];
    hipblasLtEpilogue_t reluEp = HIPBLASLT_EPILOGUE_RELU;
    hipblasOperation_t transOp = HIPBLAS_OP_T;
    
    initializeSeeds(&seed1, &seed2);
    checkCublasStatus(hipblasLtCreate(&ltHandle));
    checkCublasStatus(hipblasLtMatmulDescCreate(&OutputOpDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescCreate(&InputGradOpDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescCreate(&WeightGradOpDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(OutputOpDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &reluEp, sizeof(reluEp)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(InputGradOpDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transOp, sizeof(transOp)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(WeightGradOpDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transOp, sizeof(transOp)));
    
    checkCudaStatus(hipMalloc(&outputTensors[0], networkParameters[0] * networkBatches * sizeof(float)));
    checkCudaStatus(hipMalloc(&outputGradTensors[0], networkParameters[0] * networkBatches * sizeof(float)));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&outputLayouts[0], HIP_R_32F, networkParameters[0], networkBatches, networkParameters[0]));
    for (uint8_t i = 0; i < networkLayers; i++) {
        checkCudaStatus(hipMalloc(&weightTensors[i], networkParameters[i + 1] * networkParameters[i] * sizeof(float)));
        checkCudaStatus(hipMalloc(&outputTensors[i + 1], networkParameters[i + 1] * networkBatches * sizeof(float)));
        
        checkCudaStatus(hipMalloc(&outputGradTensors[i + 1], networkParameters[i + 1] * networkBatches * sizeof(float)));
        checkCudaStatus(hipMalloc(&weightGradTensors[i], networkParameters[i + 1] * networkParameters[i] * sizeof(float)));
        
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&weightLayouts[i], HIP_R_32F, networkParameters[i + 1], networkParameters[i], networkParameters[i + 1]));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&outputLayouts[i + 1], HIP_R_32F, networkParameters[i + 1], networkBatches, networkParameters[i + 1]));
        
        hipblasLtMatmulHeuristicResult_t heuristicResult;
        int returnedResults;
        checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, OutputOpDesc, weightLayouts[i], outputLayouts[i], outputLayouts[i + 1], outputLayouts[i + 1], preference, 1, &heuristicResult, &returnedResults));
        outputAlgos[i] = heuristicResult.algo;
        checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, InputGradOpDesc, weightLayouts[i], outputLayouts[i + 1], outputLayouts[i], outputLayouts[i], preference, 1, &heuristicResult, &returnedResults));
        inputGradAlgos[i] = heuristicResult.algo;
        checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, WeightGradOpDesc, outputLayouts[i + 1], outputLayouts[i], weightLayouts[i], weightLayouts[i], preference, 1, &heuristicResult, &returnedResults));
        weightGradAlgos[i] = heuristicResult.algo;
        
        fillDTensor(weightTensors[i], networkParameters[i + 1] * networkParameters[i], &seed1, &seed2);
    }
    
    float inputTensor[networkParameters[0] * networkBatches];
    float outputGradTensor[networkParameters[networkLayers] * networkBatches];
    float expectedTensor[networkParameters[networkLayers] * networkBatches];
    float one = 1.0f, zero = 0.0f, negOne = -1.0f, learningRate = 0.1f / networkBatches;
    for (uint32_t epoch = 0; epoch < 1000; epoch++) {
        for (uint16_t i = 0; i < networkBatches; i++) {
            mixSeed(&seed1, &seed2);
            uint8_t a = seed1 & 1, b = seed2 & 1;
            inputTensor[i * networkParameters[0]] = a;
            inputTensor[i * networkParameters[0] + 1] = b;
            inputTensor[i * networkParameters[0] + 2] = 1;
            expectedTensor[i * networkParameters[networkLayers]] = a ^ b;
        }
        
        checkCudaStatus(hipMemcpy(outputTensors[0], inputTensor, networkParameters[0] * networkBatches * sizeof(float), hipMemcpyHostToDevice));
        checkCudaStatus(hipMemcpy(outputGradTensors[networkLayers], expectedTensor, networkParameters[networkLayers] * networkBatches * sizeof(float), hipMemcpyHostToDevice));
        
        for (uint8_t i = 0; i < networkLayers; i++) {
            checkCublasStatus(hipblasLtMatmul(
                ltHandle, OutputOpDesc,
                &one,
                weightTensors[i], weightLayouts[i],
                outputTensors[i], outputLayouts[i],
                &zero,
                outputTensors[i + 1], outputLayouts[i + 1],
                outputTensors[i + 1], outputLayouts[i + 1],
                &outputAlgos[i], NULL, 0, 0));
        }
        
        addDTensors(negOne, outputGradTensors[networkLayers], outputTensors[networkLayers], networkParameters[networkLayers] * networkBatches);
        checkCudaStatus(hipMemcpy(outputGradTensor, outputGradTensors[networkLayers], networkParameters[networkLayers] * networkBatches * sizeof(float), hipMemcpyDeviceToHost));
        float error = 0;
        for (uint16_t i = networkParameters[networkLayers] * networkBatches; i--;) {
            error += fabs(outputGradTensor[i]);
        }
        printf("Error: %f\n", error / networkBatches);
        
        for (uint8_t i = networkLayers; i--;) {
            checkCublasStatus(hipblasLtMatmul(
                ltHandle, InputGradOpDesc,
                &one,
                weightTensors[i], weightLayouts[i],
                outputGradTensors[i + 1], outputLayouts[i + 1],
                &zero,
                outputGradTensors[i], outputLayouts[i],
                outputGradTensors[i], outputLayouts[i],
                &inputGradAlgos[i], NULL, 0, 0));
                
            checkCublasStatus(hipblasLtMatmul(
                ltHandle, WeightGradOpDesc,
                &one,
                outputGradTensors[i + 1], outputLayouts[i + 1],
                outputTensors[i], outputLayouts[i],
                &zero,
                weightGradTensors[i], weightLayouts[i],
                weightGradTensors[i], weightLayouts[i],
                &weightGradAlgos[i], NULL, 0, 0));
        }
        
        for (uint8_t i = 0; i < networkLayers; i++) {
            addDTensors(learningRate, weightTensors[i], weightGradTensors[i], networkParameters[i + 1] * networkParameters[i]);
        }
    }
        
    // printDTensor(outputTensors[0], networkParameters[0], networkBatches, "Input");
    // for (uint8_t i = 0; i < networkLayers; i++) {
    //     printDTensor(weightTensors[i], networkParameters[i + 1], networkParameters[i], "Weights");
    //     printDTensor(outputTensors[i + 1], networkParameters[i + 1], networkBatches, "Output");
    // }
        
    // printDTensor(outputGradTensors[networkLayers], networkParameters[networkLayers], networkBatches, "Output Grad");
    // for (uint8_t i = networkLayers; i--;) {
    //     printDTensor(weightGradTensors[i], networkParameters[i + 1], networkParameters[i], "Weight Grad");
    //     printDTensor(outputGradTensors[i], networkParameters[i], networkBatches, "Input Grad");
    // }
    
    return 0;
}