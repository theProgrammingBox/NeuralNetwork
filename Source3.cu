
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

void initializeSeeds(uint32_t *seed1, uint32_t *seed2) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    *seed1 = tv.tv_sec;
    *seed2 = tv.tv_usec;
    for (uint8_t i = 8; i--;) {
        *seed2 *= 0xbf324c81;
        *seed1 ^= *seed2;
        *seed1 *= 0x9c7493ad;
        *seed2 ^= *seed1;
    }
}

__global__ void fillDTensor(float *dTensor, uint32_t size, uint32_t seed1, uint32_t seed2) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
	seed1 ^= idx;
    seed1 *= 0x4ba1bb47;
    seed1 ^= seed2;
    seed1 *= 0xb7ebcb79;
    dTensor[idx] = (int32_t)seed1 * 0.0000000004656612875245797f;
}

void fillDTensor(float *dTensor, uint32_t size, uint32_t *seed1, uint32_t *seed2) {
    *seed2 *= 0xbf324c81;
    *seed1 ^= *seed2;
    *seed1 *= 0x9c7493ad;
    *seed2 ^= *seed1;
    fillDTensor<<<(size >> 10) + (size & 0x3ff), 0x400>>>(dTensor, size, *seed1, *seed2);
}

void printDTensor(float *dTensor, uint32_t width, uint32_t height, const char *label) {
    float *tensor = (float *)malloc(width * height * sizeof(float));
    checkCudaStatus(hipMemcpy(tensor, dTensor, width * height * sizeof(float), hipMemcpyDeviceToHost));
    printf("%s:\n", label);
    for (uint32_t i = 0; i < height; i++) {
        for (uint32_t j = 0; j < width; j++) {
            printf("%f ", tensor[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
    free(tensor);
}

int main() {
    const uint8_t networkBatches = 16;
    const uint8_t networkLayers = 2;
    uint8_t networkParameters[networkLayers + 1] = {2, 8, 1};
    float* weightTensors[networkLayers], * outputTensors[networkLayers + 1];
    float* outputGradTensors[networkLayers + 1], * weightGradTensors[networkLayers];
    uint32_t seed1, seed2;
    hipblasLtHandle_t ltHandle;
    hipblasLtMatmulDesc_t OutputOpDesc, InputGradOpDesc, WeightGradOpDesc;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatrixLayout_t weightLayouts[networkLayers], outputLayouts[networkLayers + 1];
    hipblasLtMatmulAlgo_t outputAlgos[networkLayers], inputGradAlgos[networkLayers], weightGradAlgos[networkLayers];
    hipblasLtEpilogue_t reluEp = HIPBLASLT_EPILOGUE_RELU;
    hipblasOperation_t transOp = HIPBLAS_OP_T;
    
    initializeSeeds(&seed1, &seed2);
    checkCublasStatus(hipblasLtCreate(&ltHandle));
    checkCublasStatus(hipblasLtMatmulDescCreate(&OutputOpDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescCreate(&InputGradOpDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescCreate(&WeightGradOpDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(OutputOpDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &reluEp, sizeof(reluEp)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(InputGradOpDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transOp, sizeof(transOp)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(WeightGradOpDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transOp, sizeof(transOp)));
    
    checkCudaStatus(hipMalloc(&outputTensors[0], networkParameters[0] * networkBatches * sizeof(float)));
    checkCudaStatus(hipMalloc(&outputGradTensors[0], networkParameters[0] * networkBatches * sizeof(float)));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&outputLayouts[0], HIP_R_32F, networkParameters[0], networkBatches, networkParameters[0]));
    for (uint8_t i = 0; i < networkLayers; i++) {
        checkCudaStatus(hipMalloc(&weightTensors[i], networkParameters[i + 1] * networkParameters[i] * sizeof(float)));
        checkCudaStatus(hipMalloc(&outputTensors[i + 1], networkParameters[i + 1] * networkBatches * sizeof(float)));
        
        checkCudaStatus(hipMalloc(&outputGradTensors[i + 1], networkParameters[i + 1] * networkBatches * sizeof(float)));
        checkCudaStatus(hipMalloc(&weightGradTensors[i], networkParameters[i + 1] * networkParameters[i] * sizeof(float)));
        
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&weightLayouts[i], HIP_R_32F, networkParameters[i + 1], networkParameters[i], networkParameters[i + 1]));
        checkCublasStatus(hipblasLtMatrixLayoutCreate(&outputLayouts[i + 1], HIP_R_32F, networkParameters[i + 1], networkBatches, networkParameters[i + 1]));
        
        hipblasLtMatmulHeuristicResult_t heuristicResult;
        int returnedResults;
        checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, OutputOpDesc, weightLayouts[i], outputLayouts[i], outputLayouts[i + 1], outputLayouts[i + 1], preference, 1, &heuristicResult, &returnedResults));
        outputAlgos[i] = heuristicResult.algo;
        checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, InputGradOpDesc, weightLayouts[i], outputLayouts[i + 1], outputLayouts[i], outputLayouts[i], preference, 1, &heuristicResult, &returnedResults));
        inputGradAlgos[i] = heuristicResult.algo;
        checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, WeightGradOpDesc, outputLayouts[i + 1], outputLayouts[i], weightLayouts[i], weightLayouts[i], preference, 1, &heuristicResult, &returnedResults));
        weightGradAlgos[i] = heuristicResult.algo;
        
        fillDTensor(weightTensors[i], networkParameters[i + 1] * networkParameters[i], &seed1, &seed2);
    }
    
    fillDTensor(outputTensors[0], networkParameters[0] * networkBatches, &seed1, &seed2);
    fillDTensor(outputGradTensors[networkLayers], networkParameters[networkLayers] * networkBatches, &seed1, &seed2);
    
    float alpha = 1.0f, beta = 0.0f;
    for (uint8_t i = 0; i < networkLayers; i++) {
        checkCublasStatus(hipblasLtMatmul(
            ltHandle, OutputOpDesc,
            &alpha,
            weightTensors[i], weightLayouts[i],
            outputTensors[i], outputLayouts[i],
            &beta,
            outputTensors[i + 1], outputLayouts[i + 1],
            outputTensors[i + 1], outputLayouts[i + 1],
            &outputAlgos[i], NULL, 0, 0));
    }
    
    // printDTensor(outputTensors[0], networkParameters[0], networkBatches, "Input");
    // for (uint8_t i = 0; i < networkLayers; i++) {
    //     printDTensor(weightTensors[i], networkParameters[i + 1], networkParameters[i], "Weights");
    //     printDTensor(outputTensors[i + 1], networkParameters[i + 1], networkBatches, "Output");
    // }
    
    for (uint8_t i = networkLayers; i--;) {
        checkCublasStatus(hipblasLtMatmul(
            ltHandle, InputGradOpDesc,
            &alpha,
            weightTensors[i], weightLayouts[i],
            outputGradTensors[i + 1], outputLayouts[i + 1],
            &beta,
            outputGradTensors[i], outputLayouts[i],
            outputGradTensors[i], outputLayouts[i],
            &inputGradAlgos[i], NULL, 0, 0));
            
        checkCublasStatus(hipblasLtMatmul(
            ltHandle, WeightGradOpDesc,
            &alpha,
            outputGradTensors[i + 1], outputLayouts[i + 1],
            outputTensors[i], outputLayouts[i],
            &beta,
            weightGradTensors[i], weightLayouts[i],
            weightGradTensors[i], weightLayouts[i],
            &weightGradAlgos[i], NULL, 0, 0));
    }
    
    printDTensor(outputGradTensors[networkLayers], networkParameters[networkLayers], networkBatches, "Output Grad");
    for (uint8_t i = networkLayers; i--;) {
        printDTensor(weightGradTensors[i], networkParameters[i + 1], networkParameters[i], "Weight Grad");
        printDTensor(outputGradTensors[i], networkParameters[i], networkBatches, "Input Grad");
    }
    
    return 0;
}