#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>

int main(int argc, char **argv) {
    struct timeval start, end;
    
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    float elapsedTime;
    
    gettimeofday(&start, NULL);
    hipEventRecord(start_event, 0);
    
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("Number of CUDA devices: %d\n", deviceCount);
    for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Device %d has compute capability %d.%d.\n", dev, deviceProp.major, deviceProp.minor);
    }
  
    gettimeofday(&end, NULL);
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsedTime, start_event, stop_event);
    
    printf("Cpu Time: %f ms\n", (end.tv_sec - start.tv_sec) * 1000.0 + (end.tv_usec - start.tv_usec) / 1000.0);
    printf("Gpu Time: %f ms\n", elapsedTime);
    
  return 0;
}