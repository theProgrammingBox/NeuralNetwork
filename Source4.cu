#include "hip/hip_runtime.h"
#include "Header.cuh"

struct network {
  uint32_t batchSize;
  uint32_t layers;
  uint32_t* parameters;
  float** outputs;
  float** weights;
  float** outputGradients;
  float** weightGradients;
};

void initializeNetwork(network* net, const uint32_t batchSize, const uint32_t layers, const uint32_t* parameters, uint32_t *seed1, uint32_t *seed2, bool debug = false) {
  net->batchSize = batchSize;
  net->layers = layers;
  net->parameters = (uint32_t*)malloc((net->layers + 2) * sizeof(uint32_t));
  memcpy(net->parameters, parameters, (net->layers + 2) * sizeof(uint32_t));
  
  net->outputs = (float**)malloc((net->layers + 2) * sizeof(float*));
  net->outputGradients = (float**)malloc((net->layers + 2) * sizeof(float*));
  
  if (debug) printf("Initialize network:\n");
  for (uint32_t i = 0; i < net->layers + 2; i++) {
    checkCudaStatus(hipMalloc(&net->outputs[i], net->parameters[i] * net->batchSize * sizeof(float)));
    if (debug) printDTensor(net->outputs[i], net->parameters[i], net->batchSize, "output");
    
    checkCudaStatus(hipMalloc(&net->outputGradients[i], net->parameters[i] * net->batchSize * sizeof(float)));
    if (debug) printDTensor(net->outputGradients[i], net->parameters[i], net->batchSize, "output gradient");
  }
  
  net->weights = (float**)malloc((net->layers + 1) * sizeof(float*));
  net->weightGradients = (float**)malloc((net->layers + 1) * sizeof(float*));
  
  for (uint32_t i = 0; i < net->layers + 1; i++) {
    checkCudaStatus(hipMalloc(&net->weights[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    fillDTensor(net->weights[i], net->parameters[i + 1] * net->parameters[i], seed1, seed2);
    if (debug) printDTensor(net->weights[i], net->parameters[i + 1], net->parameters[i], "weight");
    
    checkCudaStatus(hipMalloc(&net->weightGradients[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    if (debug) printDTensor(net->weightGradients[i], net->parameters[i + 1], net->parameters[i], "weight gradient");
  }
  if (debug) printf("\n");
}

void setRandomInput(network* net, uint32_t *seed1, uint32_t *seed2) {
    fillDTensor(net->outputs[0], net->parameters[0] * net->batchSize, seed1, seed2);
}

void setInput(network* net, float* inputs, bool host = true) {
  checkCudaStatus(hipMemcpy(net->outputs[0], inputs, net->parameters[0] * net->batchSize * sizeof(float), host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}

void forwardPropagate(hipblasHandle_t *cublasHandle, network* net, bool debug = false) {
  const float zero = 0.0f;
  if (debug) printf("Forward propagation:\n");
  if (debug) printDTensor(net->outputs[0], net->parameters[0], net->batchSize, "input");
  
  for (uint32_t i = 0; i < net->layers; i++) {
    float alpha = 2.0f / sqrtf(net->parameters[i]);
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
      net->parameters[i + 1], net->batchSize, net->parameters[i],
      &alpha,
      net->weights[i], net->parameters[i + 1],
      net->outputs[i], net->parameters[i],
      &zero,
      net->outputs[i + 1], net->parameters[i + 1]));
    if (debug) printDTensor(net->outputs[i + 1], net->parameters[i + 1], net->batchSize, "sum");
    
    reluForward(net->outputs[i + 1], net->parameters[i + 1] * net->batchSize);
    if (debug) printDTensor(net->outputs[i + 1], net->parameters[i + 1], net->batchSize, "relu");
  }
  
  float alpha = 2.0f / sqrtf(net->parameters[net->layers]);
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    net->parameters[net->layers + 1], net->batchSize, net->parameters[net->layers],
    &alpha,
    net->weights[net->layers], net->parameters[net->layers + 1],
    net->outputs[net->layers], net->parameters[net->layers],
    &zero,
    net->outputs[net->layers + 1], net->parameters[net->layers + 1]));
  if (debug) printDTensor(net->outputs[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output");
  
  if (debug) printf("\n");
}

void setOutputTarget(hipblasHandle_t *cublasHandle, network* net, float* target, bool debug = false) {
  checkCudaStatus(hipMemcpy(net->outputGradients[net->layers + 1], target, net->parameters[net->layers + 1] * net->batchSize * sizeof(float), hipMemcpyHostToDevice));
  const float negativeOne = -1.0f;
  checkCublasStatus(hipblasSaxpy(
    *cublasHandle,
    net->parameters[net->layers + 1] * net->batchSize,
    &negativeOne,
    net->outputs[net->layers + 1], 1,
    net->outputGradients[net->layers + 1], 1));
  if (debug) printDTensor(net->outputGradients[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output gradient");
}

void setOutputGradients(network* net, float* gradients, bool host = true) {
  checkCudaStatus(hipMemcpy(net->outputGradients[net->layers + 1], gradients, net->parameters[net->layers + 1] * net->batchSize * sizeof(float), host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}

void backPropagate(hipblasHandle_t *cublasHandle, network* net, bool errorPrint = false, bool debug = false) {
  if (errorPrint) {
    float error = 0.0f;
    checkCublasStatus(hipblasSasum(
      *cublasHandle,
      net->parameters[net->layers + 1] * net->batchSize,
      net->outputGradients[net->layers + 1], 1,
      &error));
    printf("Error: %f\n", error / net->batchSize);
  }
  
  const float zero = 0.0f;
  if (debug) printf("Back propagation:\n");
  
  float alpha = 2.0f / sqrtf(net->batchSize);
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
    net->parameters[net->layers + 1], net->parameters[net->layers], net->batchSize,
    &alpha,
    net->outputGradients[net->layers + 1], net->parameters[net->layers + 1],
    net->outputs[net->layers], net->parameters[net->layers],
    &zero,
    net->weightGradients[net->layers], net->parameters[net->layers + 1]));
  if (debug) printDTensor(net->weightGradients[net->layers], net->parameters[net->layers + 1], net->parameters[net->layers], "weight gradient");
  
  float beta = 2.0f / sqrtf(net->parameters[net->layers + 1]);
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
    net->parameters[net->layers], net->batchSize, net->parameters[net->layers + 1],
    &beta,
    net->weights[net->layers], net->parameters[net->layers + 1],
    net->outputGradients[net->layers + 1], net->parameters[net->layers + 1],
    &zero,
    net->outputGradients[net->layers], net->parameters[net->layers]));
  if (debug) printDTensor(net->outputGradients[net->layers], net->parameters[net->layers], net->batchSize, "output gradient");
  
  for (uint32_t i = net->layers; i > 0; i--) {
    reluBackward(net->outputs[i], net->outputGradients[i], net->parameters[i] * net->batchSize);
    if (debug) printDTensor(net->outputGradients[i], net->parameters[i], net->batchSize, "relu gradient");
    
    alpha = 2.0f / sqrtf(net->batchSize);
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      net->parameters[i], net->parameters[i - 1], net->batchSize,
      &alpha,
      net->outputGradients[i], net->parameters[i],
      net->outputs[i - 1], net->parameters[i - 1],
      &zero,
      net->weightGradients[i - 1], net->parameters[i]));
    if (debug) printDTensor(net->weightGradients[i - 1], net->parameters[i], net->parameters[i - 1], "weight gradient");
    
    beta = 2.0f / sqrtf(net->parameters[i]);
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      net->parameters[i - 1], net->batchSize, net->parameters[i],
      &beta,
      net->weights[i - 1], net->parameters[i],
      net->outputGradients[i], net->parameters[i],
      &zero,
      net->outputGradients[i - 1], net->parameters[i - 1]));
    if (debug) printDTensor(net->outputGradients[i - 1], net->parameters[i - 1], net->batchSize, "output gradient");
  }
  if (debug) printf("\n");
}

void updateWeights(hipblasHandle_t *cublasHandle, network* net, float learningRate, bool debug = false) {
  if (debug) printf("Update weights:\n");
  for (uint32_t i = 0; i < net->layers + 1; i++) {
    checkCublasStatus(hipblasSaxpy(
      *cublasHandle,
      net->parameters[i + 1] * net->parameters[i],
      &learningRate,
      net->weightGradients[i], 1,
      net->weights[i], 1));
    if (debug) printDTensor(net->weights[i], net->parameters[i + 1], net->parameters[i], "weight");
  }
  if (debug) printf("\n");
}

void freeNetwork(network* net, bool debug = false) {
  if (debug) printf("Free network\n");
  for (uint32_t i = 0; i < net->layers; i++)
    checkCudaStatus(hipFree(net->weights[i]));
  
  free(net->weights);
  
  for (uint32_t i = 0; i < net->layers + 1; i++)
    checkCudaStatus(hipFree(net->outputs[i]));
  
  free(net->outputs);
  
  free(net->parameters);
}

struct player {
  uint32_t idx;
  int32_t score;
};

int comparePlayers(const void *a, const void *b) {
  player *playerA = (player *)a;
  player *playerB = (player *)b;
  return playerA->score - playerB->score;
}

int main() {
  uint32_t seed1, seed2;
  initializeSeeds(&seed1, &seed2);
  
  hipblasHandle_t handle;
  checkCublasStatus(hipblasCreate(&handle));
  
  const float learningRate = 0.001f;
  const uint32_t epochs = 100;
  const uint32_t batchSize = 1024;
  
  
  network policy;
  const uint32_t policyParameters[] = {8, 16, 16, 3};
  const uint32_t policyLayers = sizeof(policyParameters) / sizeof(uint32_t) - 2;
  initializeNetwork(&policy, batchSize, policyLayers, policyParameters, &seed1, &seed2);
  
  network value;
  const uint32_t valueParameters[] = {3, 16, 16, 16, 16, 1};
  const uint32_t valueLayers = sizeof(valueParameters) / sizeof(uint32_t) - 2;
  initializeNetwork(&value, batchSize, valueLayers, valueParameters, &seed1, &seed2);
  
  float policyOutput[policyParameters[policyLayers + 1] * batchSize];
  float valueTarget[policyParameters[policyLayers + 1] * batchSize];
  float valueGradient[policyParameters[policyLayers + 1] * batchSize];
  uint32_t actions[batchSize];
  const uint32_t samples = 64;
  const int outcomes[9] = {
    0, -1,  1,
    1,  0, -1,
    -1,  1,  0
  };
  
  for (uint32_t epoch = 0; epoch < epochs; epoch++) {
    setRandomInput(&policy, &seed1, &seed2);
    forwardPropagate(&handle, &policy);
    
    setInput(&value, policy.outputs[policyLayers + 1], false);
    forwardPropagate(&handle, &value);
    
    checkCudaStatus(hipMemcpy(policyOutput, policy.outputs[policyLayers + 1], policyParameters[policyLayers + 1] * batchSize * sizeof(float), hipMemcpyDeviceToHost));
    for (uint32_t batch = 0; batch < batchSize; batch++) {
      uint32_t action = 0;
      float max = policyOutput[batch * policyParameters[policyLayers + 1]];
      for (uint32_t i = 1; i < policyParameters[policyLayers + 1]; i++) {
        if (policyOutput[batch * policyParameters[policyLayers + 1] + i] > max) {
          max = policyOutput[batch * policyParameters[policyLayers + 1] + i];
          action = i;
        }
      }
      actions[batch] = action;
    }
    
    player players[batchSize];
    for (uint32_t batch = 0; batch < batchSize; batch++) {
      int32_t score = 0;
      uint32_t action = actions[batch];
      for (uint32_t sample = 0; sample < samples; sample++) {
        mixSeed(&seed1, &seed2);
        uint32_t opponentAction = actions[seed1 % batchSize];
        score += outcomes[action * 3 + opponentAction];
      }
      
      players[batch].idx = batch;
      players[batch].score = score;
    }
    
    qsort(players, batchSize, sizeof(player), comparePlayers);
    for (uint32_t batch = 0; batch < batchSize; batch++) {
      valueTarget[players[batch].idx * valueParameters[valueLayers + 1]] = (float)batch / (batchSize - 1);
    }
    
    setOutputTarget(&handle, &value, valueTarget);
    backPropagate(&handle, &value, epoch % 100 == 0);
    updateWeights(&handle, &value, learningRate);
    
    for (uint32_t batch = 0; batch < batchSize; batch++) {
      valueGradient[players[batch].idx * valueParameters[valueLayers + 1]] = 1.0f;
    }
    setOutputGradients(&value, valueGradient);
    backPropagate(&handle, &value);
    setOutputGradients(&policy, value.outputGradients[0], false);
    backPropagate(&handle, &policy);
    updateWeights(&handle, &policy, learningRate);
  }
  freeNetwork(&policy);
  freeNetwork(&value);
  
  
  checkCublasStatus(hipblasDestroy(handle));

  return 0;
}