#include "hip/hip_runtime.h"
#include "Header.cuh"

struct network {
  uint32_t batchSize;
  uint32_t layers;
  uint32_t* parameters;
  float** outputs;
  float** weights;
  float** outputGradients;
  float** weightGradients;
};

void initializeNetwork(network* net, const uint32_t batchSize, const uint32_t layers, const uint32_t* parameters, uint32_t *seed1, uint32_t *seed2, bool debug = false) {
  net->batchSize = batchSize;
  net->layers = layers;
  net->parameters = (uint32_t*)malloc((net->layers + 2) * sizeof(uint32_t));
  memcpy(net->parameters, parameters, (net->layers + 2) * sizeof(uint32_t));
  
  net->outputs = (float**)malloc((net->layers + 2) * sizeof(float*));
  net->outputGradients = (float**)malloc((net->layers + 2) * sizeof(float*));
  
  if (debug) printf("Initialize network:\n");
  for (uint32_t i = 0; i < net->layers + 2; i++) {
    checkCudaStatus(hipMalloc(&net->outputs[i], net->parameters[i] * net->batchSize * sizeof(float)));
    if (debug) printDTensor(net->outputs[i], net->parameters[i], net->batchSize, "output");
    
    checkCudaStatus(hipMalloc(&net->outputGradients[i], net->parameters[i] * net->batchSize * sizeof(float)));
    if (debug) printDTensor(net->outputGradients[i], net->parameters[i], net->batchSize, "output gradient");
  }
  
  net->weights = (float**)malloc((net->layers + 1) * sizeof(float*));
  net->weightGradients = (float**)malloc((net->layers + 1) * sizeof(float*));
  
  for (uint32_t i = 0; i < net->layers + 1; i++) {
    checkCudaStatus(hipMalloc(&net->weights[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    fillDTensor(net->weights[i], net->parameters[i + 1] * net->parameters[i], seed1, seed2);
    if (debug) printDTensor(net->weights[i], net->parameters[i + 1], net->parameters[i], "weight");
    
    checkCudaStatus(hipMalloc(&net->weightGradients[i], net->parameters[i + 1] * net->parameters[i] * sizeof(float)));
    if (debug) printDTensor(net->weightGradients[i], net->parameters[i + 1], net->parameters[i], "weight gradient");
  }
  if (debug) printf("\n");
}

void setRandomInput(network* net, uint32_t *seed1, uint32_t *seed2) {
    fillDTensor(net->outputs[0], net->parameters[0] * net->batchSize, seed1, seed2);
}

void setCustomRandomInput(network* net, uint32_t *seed1, uint32_t *seed2) {
  customFillDTensor(net->outputs[0], net->parameters[0] * net->batchSize, seed1, seed2);
}

void setInput(network* net, float* inputs, bool host = true) {
  checkCudaStatus(hipMemcpy(net->outputs[0], inputs, net->parameters[0] * net->batchSize * sizeof(float), host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}

void forwardPropagate(hipblasHandle_t *cublasHandle, network* net, bool debug = false) {
  const float zero = 0.0f;
  if (debug) printf("Forward propagation:\n");
  if (debug) printDTensor(net->outputs[0], net->parameters[0], net->batchSize, "input");
  
  for (uint32_t i = 0; i < net->layers; i++) {
    // float alpha = 2.0f / sqrtf(net->parameters[i]);
    float alpha = 1.0f;
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
      net->parameters[i + 1], net->batchSize, net->parameters[i],
      &alpha,
      net->weights[i], net->parameters[i + 1],
      net->outputs[i], net->parameters[i],
      &zero,
      net->outputs[i + 1], net->parameters[i + 1]));
    if (debug) printDTensor(net->outputs[i + 1], net->parameters[i + 1], net->batchSize, "sum");
    
    reluForward(net->outputs[i + 1], net->parameters[i + 1] * net->batchSize);
    if (debug) printDTensor(net->outputs[i + 1], net->parameters[i + 1], net->batchSize, "relu");
  }
  
  // float alpha = 2.0f / sqrtf(net->parameters[net->layers]);
  float alpha = 1.0f;
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    net->parameters[net->layers + 1], net->batchSize, net->parameters[net->layers],
    &alpha,
    net->weights[net->layers], net->parameters[net->layers + 1],
    net->outputs[net->layers], net->parameters[net->layers],
    &zero,
    net->outputs[net->layers + 1], net->parameters[net->layers + 1]));
  if (debug) printDTensor(net->outputs[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output");
  
  if (debug) printf("\n");
}

void setOutputTarget(hipblasHandle_t *cublasHandle, network* net, float* target, bool debug = false) {
  checkCudaStatus(hipMemcpy(net->outputGradients[net->layers + 1], target, net->parameters[net->layers + 1] * net->batchSize * sizeof(float), hipMemcpyHostToDevice));
  const float negativeOne = -1.0f;
  checkCublasStatus(hipblasSaxpy(
    *cublasHandle,
    net->parameters[net->layers + 1] * net->batchSize,
    &negativeOne,
    net->outputs[net->layers + 1], 1,
    net->outputGradients[net->layers + 1], 1));
  if (debug) printDTensor(net->outputGradients[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output gradient");
}

void setOutputGradients(network* net, float* gradients, bool host = true) {
  checkCudaStatus(hipMemcpy(net->outputGradients[net->layers + 1], gradients, net->parameters[net->layers + 1] * net->batchSize * sizeof(float), host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice));
}

void backPropagate(hipblasHandle_t *cublasHandle, network* net, bool errorPrint = false, bool debug = false) {
  if (errorPrint) {
    float error = 0.0f;
    checkCublasStatus(hipblasSasum(
      *cublasHandle,
      net->parameters[net->layers + 1] * net->batchSize,
      net->outputGradients[net->layers + 1], 1,
      &error));
    printf("Error: %f\n", error / net->batchSize);
  }
  
  const float zero = 0.0f;
  if (debug) printf("Back propagation:\n");
  
  if (debug) printDTensor(net->outputGradients[net->layers + 1], net->parameters[net->layers + 1], net->batchSize, "output gradient");
  
  // float alpha = 2.0f / sqrtf(net->batchSize);
  float alpha = 1.0f;
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
    net->parameters[net->layers + 1], net->parameters[net->layers], net->batchSize,
    &alpha,
    net->outputGradients[net->layers + 1], net->parameters[net->layers + 1],
    net->outputs[net->layers], net->parameters[net->layers],
    &zero,
    net->weightGradients[net->layers], net->parameters[net->layers + 1]));
  if (debug) printDTensor(net->weightGradients[net->layers], net->parameters[net->layers + 1], net->parameters[net->layers], "weight gradient");
  
  // float beta = 2.0f / sqrtf(net->parameters[net->layers + 1]);
  float beta = 1.0f;
  checkCublasStatus(hipblasSgemm(
    *cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
    net->parameters[net->layers], net->batchSize, net->parameters[net->layers + 1],
    &beta,
    net->weights[net->layers], net->parameters[net->layers + 1],
    net->outputGradients[net->layers + 1], net->parameters[net->layers + 1],
    &zero,
    net->outputGradients[net->layers], net->parameters[net->layers]));
  if (debug) printDTensor(net->outputGradients[net->layers], net->parameters[net->layers], net->batchSize, "output gradient");
  
  for (uint32_t i = net->layers; i > 0; i--) {
    reluBackward(net->outputs[i], net->outputGradients[i], net->parameters[i] * net->batchSize);
    if (debug) printDTensor(net->outputGradients[i], net->parameters[i], net->batchSize, "relu gradient");
    
    // alpha = 2.0f / sqrtf(net->batchSize);
    alpha = 1.0f;
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T,
      net->parameters[i], net->parameters[i - 1], net->batchSize,
      &alpha,
      net->outputGradients[i], net->parameters[i],
      net->outputs[i - 1], net->parameters[i - 1],
      &zero,
      net->weightGradients[i - 1], net->parameters[i]));
    if (debug) printDTensor(net->weightGradients[i - 1], net->parameters[i], net->parameters[i - 1], "weight gradient");
    
    // beta = 2.0f / sqrtf(net->parameters[i]);
    beta = 1.0f;
    checkCublasStatus(hipblasSgemm(
      *cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N,
      net->parameters[i - 1], net->batchSize, net->parameters[i],
      &beta,
      net->weights[i - 1], net->parameters[i],
      net->outputGradients[i], net->parameters[i],
      &zero,
      net->outputGradients[i - 1], net->parameters[i - 1]));
    if (debug) printDTensor(net->outputGradients[i - 1], net->parameters[i - 1], net->batchSize, "output gradient");
  }
  if (debug) printf("\n");
}

void updateWeights(hipblasHandle_t *cublasHandle, network* net, float learningRate, bool debug = false) {
  if (debug) printf("Update weights:\n");
  for (uint32_t i = 0; i < net->layers + 1; i++) {
    checkCublasStatus(hipblasSaxpy(
      *cublasHandle,
      net->parameters[i + 1] * net->parameters[i],
      &learningRate,
      net->weightGradients[i], 1,
      net->weights[i], 1));
    if (debug) printDTensor(net->weights[i], net->parameters[i + 1], net->parameters[i], "weight");
  }
  if (debug) printf("\n");
}

void freeNetwork(network* net, bool debug = false) {
  if (debug) printf("Free network\n");
  for (uint32_t i = 0; i < net->layers; i++)
    checkCudaStatus(hipFree(net->weights[i]));
  
  free(net->weights);
  
  for (uint32_t i = 0; i < net->layers + 1; i++)
    checkCudaStatus(hipFree(net->outputs[i]));
  
  free(net->outputs);
  
  free(net->parameters);
}

int main() {
  uint32_t seed1, seed2;
  initializeSeeds(&seed1, &seed2);
  
  hipblasHandle_t handle;
  checkCublasStatus(hipblasCreate(&handle));
  
  const float policyLearningRate = 0.00001f;
  const float valueLearningRate = 0.001f;
  const uint32_t epochs = 1000;
  const uint32_t batchSize = 256;
  
  
  network policy;
  const uint32_t policyParameters[] = {2, 8, 1};
  const uint32_t policyLayers = sizeof(policyParameters) / sizeof(uint32_t) - 2;
  initializeNetwork(&policy, batchSize, policyLayers, policyParameters, &seed1, &seed2);
  
  network value;
  const uint32_t valueParameters[] = {2, 10, 1};
  const uint32_t valueLayers = sizeof(valueParameters) / sizeof(uint32_t) - 2;
  initializeNetwork(&value, batchSize, valueLayers, valueParameters, &seed1, &seed2);
  
  float policyOutput[policyParameters[policyLayers + 1] * batchSize];
  float valueTarget[policyParameters[policyLayers + 1] * batchSize];
  float valueGradient[policyParameters[policyLayers + 1] * batchSize];
  uint32_t actions[batchSize];
  const uint32_t samples = 1024;
  const int outcomes[9] = {
    0, -1,  1,
    1,  0, -1,
    -1,  1,  0
  };
  
  float valueInput[valueParameters[0] * batchSize];
  for (uint32_t epoch = 0; epoch < epochs; epoch++) {
    for (uint32_t batch = 0; batch < batchSize; batch++) {
      valueInput[batch * 2] = generateRandomUint32(&seed1, &seed2) % 3;
      valueInput[batch * 2 + 1] = 1.0f;
      switch ((int)valueInput[batch * 2]) {
        case 0:
          valueTarget[batch] = 0.5f;
          break;
        case 1:
          valueTarget[batch] = 1.0f;
          break;
        case 2:
          valueTarget[batch] = 0.25f;
          break;
      }
    }
      
    // feedforward
    setInput(&value, valueInput);
    forwardPropagate(&handle, &value);
    
    // set target
    setOutputTarget(&handle, &value, valueTarget);
    backPropagate(&handle, &value, epoch % 100 == 0);
    
    // update weights
    updateWeights(&handle, &value, valueLearningRate);
  }
  
  // now graph inputs to outputs. 0 - 2 using 10 points. so point 1 is 0, etc
  for (uint32_t point = 0; point < 11; point++) {
    valueInput[0] = (float)point / 5.0f;
    setInput(&value, valueInput);
    forwardPropagate(&handle, &value);
    
    float valueOutput[valueParameters[valueLayers + 1] * batchSize];
    checkCudaStatus(hipMemcpy(valueOutput, value.outputs[valueLayers + 1], valueParameters[valueLayers + 1] * batchSize * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("%f %f\n", valueInput[0], valueOutput[0]);
  }
  
  freeNetwork(&policy);
  freeNetwork(&value);
  
  
  checkCublasStatus(hipblasDestroy(handle));

  return 0;
}