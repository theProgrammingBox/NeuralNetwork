
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

inline void checkCublasStatus(hipblasStatus_t status) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("CuBLAS error: %d\n", status);
    exit(1); 
  }
}

int main() {

  const int widthA = 32;
  const int heightA = 32;
  const int widthC = 64;
  
  hipblasHandle_t handle;
  checkCublasStatus(hipblasCreate(&handle));

  // Rest of code

  checkCublasStatus(hipblasDestroy(handle));

  return 0;
}