
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <hipblas.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("CuBLAS error: %d\n", status);
    exit(1);
  }
}

void initializeSeeds(uint32_t *seed1, uint32_t *seed2) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    *seed1 = tv.tv_sec;
    *seed2 = tv.tv_usec;
    for (uint8_t i = 8; i--;) {
        *seed2 *= 0xbf324c81;
        *seed1 ^= *seed2;
        *seed1 *= 0x9c7493ad;
        *seed2 ^= *seed1;
    }
}

__global__ void _fillDTensor(float *dTensor, uint32_t size, uint32_t seed1, uint32_t seed2) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
	seed1 ^= idx;
    seed1 *= 0x4ba1bb47;
    seed1 ^= seed2;
    seed1 *= 0xb7ebcb79;
    dTensor[idx] = (int32_t)seed1 * 0.0000000004656612875245797f;
}

void fillDTensor(float *dTensor, uint32_t size, uint32_t *seed1, uint32_t *seed2) {
    *seed2 *= 0xbf324c81;
    *seed1 ^= *seed2;
    *seed1 *= 0x9c7493ad;
    *seed2 ^= *seed1;
    _fillDTensor<<<(size >> 10) + (size & 0x3ff), 0x400>>>(dTensor, size, *seed1, *seed2);
}

void printDTensor(float *dTensor, uint32_t width, uint32_t height, const char *label) {
    float *tensor = (float *)malloc(width * height * sizeof(float));
    checkCudaStatus(hipMemcpy(tensor, dTensor, width * height * sizeof(float), hipMemcpyDeviceToHost));
    printf("%s:\n", label);
    for (uint32_t i = 0; i < height; i++) {
        for (uint32_t j = 0; j < width; j++) {
            printf("%f ", tensor[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
    free(tensor);
}

int compareFloats(const void* a, const void* b) {
  float fa = *(const float*) a;
  float fb = *(const float*) b;
  return (fa > fb) - (fa < fb);
}

__global__ void compareTensors(const float *tensorA, const float *tensorB, uint32_t size, bool *result) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;

    if (fabsf(tensorA[idx] - tensorB[idx]) > 0.001f) {
        *result = false;
    }
}

bool areTensorsEqual(float *dTensorA, float *dTensorB, uint32_t size) {
    bool *dResult, hResult = true;
    hipMalloc(&dResult, sizeof(bool));
    hipMemcpy(dResult, &hResult, sizeof(bool), hipMemcpyHostToDevice);

    uint32_t threadsPerBlock = 256;
    uint32_t blocks = (size + threadsPerBlock - 1) / threadsPerBlock;
    
    compareTensors<<<blocks, threadsPerBlock>>>(dTensorA, dTensorB, size, dResult);

    hipMemcpy(&hResult, dResult, sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(dResult);
    
    return hResult;
}

__global__ void matrixMul(const float *A, const float *B, float *C, 
                          int M, int N, int K) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    if(ty < M && tx < N) {
        float c = 0;
        for(int i = 0; i < K; ++i){
            c += A[ty * K + i] * B[i * N + tx];
        }
        C[ty * N + tx] = c;
    }
}

template <int BLOCK>
__global__ void sgemm(int m, int n, int k, float *a, int lda, float *b, int ldb,
                      float *c, int ldc) {
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int bx = blockIdx.x;
  const int by = blockIdx.y;

  float *begin_a = a + by * BLOCK * k;
  float *begin_b = b + bx * BLOCK;
  float *end_a = begin_a + k;

  float sum = 0.f;
  for (float *a_ptr = begin_a, *b_ptr = begin_b; a_ptr < end_a;
       a_ptr += BLOCK, b_ptr += BLOCK * n) {
    __shared__ float ashare[BLOCK][BLOCK];
    __shared__ float bshare[BLOCK][BLOCK];

    ashare[ty][tx] = a_ptr[ty * k + tx];
    bshare[ty][tx] = b_ptr[ty * n + tx];
    __syncthreads();

#pragma unroll
    for (int kk = 0; kk < BLOCK; ++kk) {
      sum += ashare[ty][kk] * bshare[kk][tx];
    }
    __syncthreads();
  }

  c[(BLOCK * by + ty) * n + BLOCK * bx + tx] = sum;
}


int main() {
  uint32_t seed1, seed2;
  initializeSeeds(&seed1, &seed2);

  const uint32_t widthA = 1024;
  const uint32_t heightA = 2048; 
  const uint32_t widthC = 512;
  
  float *dTensorA, *dTensorB, *dTensorC, *dTensorCCublas;
  
  hipblasHandle_t handle;
  checkCublasStatus(hipblasCreate(&handle));

  hipMalloc(&dTensorA, widthA * heightA * sizeof(float));
  hipMalloc(&dTensorB, widthC * widthA * sizeof(float)); 
  hipMalloc(&dTensorC, heightA * widthC * sizeof(float));
  hipMalloc(&dTensorCCublas, heightA * widthC * sizeof(float));
  
  fillDTensor(dTensorA, widthA * heightA, &seed1, &seed2);
  fillDTensor(dTensorB, widthC * widthA, &seed1, &seed2);
  
  // printDTensor(dTensorA, widthA, heightA, "A");
  // printDTensor(dTensorB, widthC, widthA, "B");
  
  const uint32_t samples = 1024 * 4;
  
  float times[samples];
  
  const float alpha = 1.0f;
  const float beta = 0.0f;
  
  struct timeval t_start, t_end;
  
  float mean, median;
  
  
  
  for (uint32_t i = samples; i--;) {

    gettimeofday(&t_start, NULL);
  
    hipblasSgemm(
      handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
      widthC, heightA, widthA,
      &alpha,
      dTensorB, widthC,
      dTensorA, widthA,
      &beta,
      dTensorCCublas, widthC);
      
    gettimeofday(&t_end, NULL);

    times[i] = (t_end.tv_sec - t_start.tv_sec) * 1000.0f + (t_end.tv_usec - t_start.tv_usec) / 1000.0f;
  }
    
  // printDTensor(dTensorCCublas, widthC, heightA, "C");
  
  mean = 0.0f;
  for (uint32_t i = samples; i--;) mean += times[i];
  mean /= samples;
  
  qsort(times, samples, sizeof(float), compareFloats);
  median = times[samples >> 1];
  
  printf("Mean: %f ms\n", mean);
  printf("Median: %f ms\n", median);
  
  
  
  for (uint32_t i = samples; i--;) {

    gettimeofday(&t_start, NULL);
    
    dim3 threads(32, 32);
    dim3 blocks((widthC + threads.x - 1) / threads.x, (heightA + threads.y - 1) / threads.y);
    matrixMul<<<blocks, threads>>>(dTensorA, dTensorB, dTensorC, heightA, widthC, widthA);
      
    gettimeofday(&t_end, NULL);

    times[i] = (t_end.tv_sec - t_start.tv_sec) * 1000.0f + (t_end.tv_usec - t_start.tv_usec) / 1000.0f;
  }
    
  areTensorsEqual(dTensorC, dTensorCCublas, heightA * widthC) ? printf("\nTensors are equal\n") : printf("\nTensors are not equal\n");
  
  mean = 0.0f;
  for (uint32_t i = samples; i--;) mean += times[i];
  mean /= samples;
  
  qsort(times, samples, sizeof(float), compareFloats);
  median = times[samples >> 1];
  
  printf("Mean: %f ms\n", mean);
  printf("Median: %f ms\n", median);
  
  
  
  for (uint32_t i = samples; i--;) {

    gettimeofday(&t_start, NULL);
    
    dim3 threads(32, 32);
    dim3 blocks((widthC + threads.x - 1) / threads.x, (heightA + threads.y - 1) / threads.y);
    sgemm<32><<<blocks, threads>>>(heightA, widthC, widthA, dTensorA, widthA, dTensorB, widthC, dTensorC, widthC);
      
    gettimeofday(&t_end, NULL);

    times[i] = (t_end.tv_sec - t_start.tv_sec) * 1000.0f + (t_end.tv_usec - t_start.tv_usec) / 1000.0f;
  }
  
  areTensorsEqual(dTensorC, dTensorCCublas, heightA * widthC) ? printf("\nTensors are equal\n") : printf("\nTensors are not equal\n");
  
  mean = 0.0f;
  for (uint32_t i = samples; i--;) mean += times[i];
  mean /= samples;
  
  qsort(times, samples, sizeof(float), compareFloats);
  median = times[samples >> 1];
  
  printf("Mean: %f ms\n", mean);
  printf("Median: %f ms\n", median);
  
  
  
  hipFree(dTensorA);
  hipFree(dTensorB);
  hipFree(dTensorC);
  hipFree(dTensorCCublas);
  
  checkCublasStatus(hipblasDestroy(handle));

  return 0;
}