#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

int main()
{
    hipblasLtHandle_t ltHandle;
    checkCublasStatus(hipblasLtCreate(&ltHandle));

    const int aWidth = 4;
    const int aHeight = 2;
    const int dWidth = 3;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    // Create descriptors
    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatrixLayout_t aDesc, bDesc, cDesc;

    checkCublasStatus(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
    
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&aDesc, HIP_R_32F, dWidth, aWidth, dWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&bDesc, HIP_R_32F, aWidth, aHeight, aWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&cDesc, HIP_R_32F, dWidth, aHeight, dWidth));

    checkCublasStatus(hipblasLtMatrixLayoutDestroy(cDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(bDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(aDesc));
    checkCublasStatus(hipblasLtMatmulDescDestroy(opDesc));
    checkCublasStatus(hipblasLtDestroy(ltHandle));

    return 0;
}