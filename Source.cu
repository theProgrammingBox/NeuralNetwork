#include <stdio.h>
// #include <stdlib.h>
// #include <time.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

int main()
{
    hipblasHandle_t handle;
    checkCublasStatus(hipblasCreate(&handle));

    float *d_A, *d_B, *d_C;
    checkCudaStatus(hipMalloc((void **)&d_A, sizeof(float) * 3 * 2));
    checkCudaStatus(hipMalloc((void **)&d_B, sizeof(float) * 2 * 3));
    checkCudaStatus(hipMalloc((void **)&d_C, sizeof(float) * 3 * 3));

    float h_A[6] = {1, 2, 3, 4, 5, 6};
    float h_B[6] = {1, 2, 3, 4, 5, 6};
    float h_C[9] = {0};

    checkCudaStatus(hipMemcpy(d_A, h_A, sizeof(float) * 3 * 2, hipMemcpyHostToDevice));
    checkCudaStatus(hipMemcpy(d_B, h_B, sizeof(float) * 2 * 3, hipMemcpyHostToDevice));
    checkCudaStatus(hipMemcpy(d_C, h_C, sizeof(float) * 3 * 3, hipMemcpyHostToDevice));

    float alpha = 1.0f;
    float beta = 0.0f;

    checkCublasStatus(hipblasSgemm(
        handle, HIPBLAS_OP_N,HIPBLAS_OP_N,
        3, 3, 2,
        &alpha,
        d_A, 3,
        d_B, 2,
        &beta,
        d_C, 3));

    checkCudaStatus(hipMemcpy(h_C, d_C, sizeof(float) * 3 * 3, hipMemcpyDeviceToHost));

    for (int i = 0; i < 9; i++)
        printf("%f ", h_C[i]);

    checkCudaStatus(hipFree(d_A));
    checkCudaStatus(hipFree(d_B));
    checkCudaStatus(hipFree(d_C));

    return 0;
}