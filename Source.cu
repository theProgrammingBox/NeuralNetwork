#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

inline void printTensorDev(float* TensorDev, int width, int height, const char* name)
{
    float* TensorHost = (float*)malloc(width * height * sizeof(float));
    checkCudaStatus(hipMemcpy(TensorHost, TensorDev, width * height * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("%s:\n", name);
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
            printf("%f, ", TensorHost[i * width + j]);
        printf("\n");
    }
    printf("\n");
    free(TensorHost);
}

int main()
{
    hipblasLtHandle_t ltHandle;
    checkCublasStatus(hipblasLtCreate(&ltHandle));

    const int aWidth = 4;
    const int aHeight = 2;
    const int dWidth = 3;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    // Create descriptors
    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatrixLayout_t aDesc, bDesc, cDesc;

    checkCublasStatus(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
    
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&aDesc, HIP_R_32F, dWidth, aWidth, dWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&bDesc, HIP_R_32F, aWidth, aHeight, aWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&cDesc, HIP_R_32F, dWidth, aHeight, dWidth));

    // heuristics
    int returnedResults = 0;
    const int requestedAlgoCount = 32;
    hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount];
    hipblasLtMatmulPreference_t preference;
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, opDesc, aDesc, bDesc, cDesc, cDesc, preference, requestedAlgoCount, heuristicResult, &returnedResults));
    
    if (returnedResults == 0)
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    
    // print heuristics
    for (int i = 0; i < returnedResults; i++)
        printf("heuristic %d: workspace %zu\n", i, heuristicResult[i].workspaceSize);
    printf("\n");
    
    // allocate memory
    float *aDev, *bDev, *cDev;
    checkCudaStatus(hipMalloc((void**)&aDev, dWidth * aWidth * sizeof(float)));
    checkCudaStatus(hipMalloc((void**)&bDev, aWidth * aHeight * sizeof(float)));
    checkCudaStatus(hipMalloc((void**)&cDev, dWidth * aHeight * sizeof(float)));
    
    // print tensors
    printTensorDev(aDev, aWidth, dWidth, "a");
    printTensorDev(bDev, aWidth, aHeight, "b");
    printTensorDev(cDev, aHeight, dWidth, "c");
    
    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(cDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(bDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(aDesc));
    checkCublasStatus(hipblasLtMatmulDescDestroy(opDesc));
    checkCublasStatus(hipblasLtDestroy(ltHandle));

    return 0;
}