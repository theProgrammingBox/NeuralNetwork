#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

int main()
{
    hipblasLtHandle_t ltHandle;
    checkCublasStatus(hipblasLtCreate(&ltHandle));

    const int aWidth = 4;
    const int aHeight = 2;
    const int dWidth = 3;

    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    // Create descriptors
    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatrixLayout_t aDesc, bDesc, cDesc;

    checkCublasStatus(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
    
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&aDesc, HIP_R_32F, dWidth, aWidth, dWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&bDesc, HIP_R_32F, aWidth, aHeight, aWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&cDesc, HIP_R_32F, dWidth, aHeight, dWidth));

    // heuristics
    hipblasLtMatmulPreference_t preference = NULL;
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    const int requestedAlgoCount = 32;
    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult[requestedAlgoCount] = { 0 };
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, opDesc, aDesc, bDesc, cDesc, cDesc, preference, requestedAlgoCount, heuristicResult, &returnedResults));
    
    if (returnedResults == 0) {
        checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);
    }
    
    // print heuristics
    for (int i = 0; i < returnedResults; i++)
        printf("heuristic %d: workspace %zu\n", i, heuristicResult[i].workspaceSize);

    checkCublasStatus(hipblasLtMatrixLayoutDestroy(cDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(bDesc));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(aDesc));
    checkCublasStatus(hipblasLtMatmulDescDestroy(opDesc));
    checkCublasStatus(hipblasLtDestroy(ltHandle));

    return 0;
}