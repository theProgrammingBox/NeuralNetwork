
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    // Tensor dimensions
    int batchSize = 1, channels = 1, height = 1, width = 10;

    // Host data for forward pass
    std::vector<float> h_input(width);
    for (int i = 0; i < width; ++i) {
        h_input[i] = i - 5; // Example input data (negative and positive values)
    }

    // Device data for forward pass
    float *d_input, *d_output;
    hipMalloc(&d_input, batchSize * channels * height * width * sizeof(float));
    hipMalloc(&d_output, batchSize * channels * height * width * sizeof(float));

    // Copy host data to device for forward pass
    hipMemcpy(d_input, h_input.data(), batchSize * channels * height * width * sizeof(float), hipMemcpyHostToDevice);

    // Tensor descriptor
    cudnnTensorDescriptor_t inOutTensorDesc;
    cudnnCreateTensorDescriptor(&inOutTensorDesc);
    cudnnSetTensor4dDescriptor(inOutTensorDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, batchSize, channels, height, width);

    // ReLU forward
    float alpha = 1.0f, beta = 0.0f;
    cudnnActivationDescriptor_t activationDesc;
    cudnnCreateActivationDescriptor(&activationDesc);
    cudnnSetActivationDescriptor(activationDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0.0);

    cudnnActivationForward(cudnn, activationDesc,
                           &alpha, inOutTensorDesc, d_input,
                           &beta, inOutTensorDesc, d_output);

    // Copy device data back to host for forward pass
    std::vector<float> h_output(width);
    hipMemcpy(h_output.data(), d_output, batchSize * channels * height * width * sizeof(float), hipMemcpyDeviceToHost);

    // Print the output tensor
    std::cout << "ReLU Output:" << std::endl;
    for (int i = 0; i < width; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Allocate memory for the gradient tensors
    float *d_input_grad, *d_output_grad;
    hipMalloc(&d_input_grad, batchSize * channels * height * width * sizeof(float));
    hipMalloc(&d_output_grad, batchSize * channels * height * width * sizeof(float));

    // Initialize the output gradient with arbitrary values
    std::vector<float> h_output_grad(width, 1.0f); // Example gradient

    // Copy host output gradient data to device
    hipMemcpy(d_output_grad, h_output_grad.data(), batchSize * channels * height * width * sizeof(float), hipMemcpyHostToDevice);

    // ReLU backward
    cudnnActivationBackward(cudnn, activationDesc,
                            &alpha, inOutTensorDesc, d_output, // ReLU output as input
                            inOutTensorDesc, d_output_grad,    // gradient w.r.t ReLU output
                            inOutTensorDesc, d_input,          // ReLU input
                            &beta, inOutTensorDesc, d_input_grad); // gradient w.r.t ReLU input

    // Copy device input gradient data back to host
    std::vector<float> h_input_grad(width);
    hipMemcpy(h_input_grad.data(), d_input_grad, batchSize * channels * height * width * sizeof(float), hipMemcpyDeviceToHost);

    // Print the input gradient tensor
    std::cout << "ReLU Gradient:" << std::endl;
    for (int i = 0; i < width; ++i) {
        std::cout << h_input_grad[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_input_grad);
    hipFree(d_output_grad);
    cudnnDestroyActivationDescriptor(activationDesc);
    cudnnDestroyTensorDescriptor(inOutTensorDesc);
    cudnnDestroy(cudnn);

    return 0;
}
