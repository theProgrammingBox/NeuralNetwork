
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

void initializeSeeds(uint32_t *seed1, uint32_t *seed2) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    *seed1 = tv.tv_sec;
    *seed2 = tv.tv_usec;
    for (uint8_t i = 8; i--;) {
        *seed2 *= 0xbf324c81;
        *seed1 ^= *seed2;
        *seed1 *= 0x9c7493ad;
        *seed2 ^= *seed1;
    }
}

__global__ void fillDTensor(float *dTensor, uint32_t size, uint32_t seed1, uint32_t seed2) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
	seed1 ^= idx;
    seed1 *= 0x4ba1bb47;
    seed1 ^= seed2;
    seed1 *= 0xb7ebcb79;
    dTensor[idx] = (int32_t)seed1 * 0.0000000004656612875245797f;
}

void fillDTensor(float *dTensor, uint32_t size, uint32_t *seed1, uint32_t *seed2) {
    *seed2 *= 0xbf324c81;
    *seed1 ^= *seed2;
    *seed1 *= 0x9c7493ad;
    *seed2 ^= *seed1;
    fillDTensor<<<(size >> 10) + (size & 0x3ff), 0x400>>>(dTensor, size, *seed1, *seed2);
}

void printDTensor(float *dTensor, uint32_t width, uint32_t height, const char *label) {
    float *tensor = (float *)malloc(width * height * sizeof(float));
    checkCudaStatus(hipMemcpy(tensor, dTensor, width * height * sizeof(float), hipMemcpyDeviceToHost));
    printf("%s:\n", label);
    for (uint32_t i = 0; i < height; i++) {
        for (uint32_t j = 0; j < width; j++) {
            printf("%f ", tensor[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
    free(tensor);
}

void printBits(void *mem, size_t size) {
    unsigned char *bytes = (unsigned char *)mem;
    for (size_t i = 0; i < size; i++) {
        for (int j = 7; j >= 0; j--) {
            printf("%c", (bytes[i] & (1 << j)) ? '1' : '0');
        }
    }
    printf("\n");
}

int main() {
    uint32_t seed1, seed2;
    initializeSeeds(&seed1, &seed2);
    
    uint16_t aWidth = 4, aHeight = 3, dWidth = 2;
    
    float *dTensorA, *dTensorB, *dTensorC, *dTensorD;
    checkCudaStatus(hipMalloc((void **)&dTensorA, dWidth * aWidth * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorB, aWidth * aHeight * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorC, dWidth * 1 * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorD, dWidth * aHeight * sizeof(float)));
    
    float *dTensorDG;
    checkCudaStatus(hipMalloc((void **)&dTensorDG, dWidth * aWidth * sizeof(float)));
    
    fillDTensor(dTensorA, dWidth * aWidth, &seed1, &seed2);
    fillDTensor(dTensorB, aWidth * aHeight, &seed1, &seed2);
    fillDTensor(dTensorC, dWidth * 1, &seed1, &seed2);
    
    printDTensor(dTensorA, dWidth, aWidth, "Weight");
    printDTensor(dTensorB, aWidth, aHeight, "Input");
    // printDTensor(dTensorC, dWidth, 1, "Bias");
    
    hipblasLtHandle_t ltHandle;
    
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_RELU_AUX;
    
    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatrixLayout_t descA, descB, descD;
    
    int returnedResults;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulAlgo_t algo;
    
    checkCublasStatus(hipblasLtCreate(&ltHandle));
    
    size_t size = 128;
    void* idk;
    checkCudaStatus(hipMalloc(&idk, size));
    void *hostMemory = malloc(size);
    
    checkCublasStatus(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &idk, sizeof(idk)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &size, sizeof(size)));
    
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&descA, HIP_R_32F, dWidth, aWidth, dWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&descB, HIP_R_32F, aWidth, aHeight, aWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&descD, HIP_R_32F, dWidth, aHeight, dWidth));
    
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, opDesc, descA, descB, descD, descD, preference, 1, &heuristicResult, &returnedResults));
    
    algo = heuristicResult.algo;
    float alpha = 1.0f, beta = 0.0f;
    
    checkCublasStatus(hipblasLtMatmul(
        ltHandle, opDesc,
        &alpha,
        dTensorA, descA,
        dTensorB, descB,
        &beta,
        dTensorD, descD,
        dTensorD, descD,
        &algo, NULL, 0, 0));
        
    printDTensor(dTensorD, dWidth, aHeight, "Output");
    
    
    checkCudaStatus(hipMemcpy(hostMemory, idk, size, hipMemcpyDeviceToHost));
    printBits(hostMemory, size);
    
    float* hTensorDGrad = (float *)malloc(dWidth * aHeight * sizeof(float));
    for (uint32_t i = 0; i < dWidth * aHeight; i++) {
        hTensorDGrad[i] = 1.0f;
    }
    
    float *dTensorAGrad, *dTensorBGrad, *dTensorDGrad;
    checkCudaStatus(hipMalloc((void **)&dTensorAGrad, dWidth * aWidth * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorBGrad, aWidth * aHeight * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorDGrad, dWidth * aHeight * sizeof(float)));
    
    checkCudaStatus(hipMemcpy(dTensorDGrad, hTensorDGrad, dWidth * aHeight * sizeof(float), hipMemcpyHostToDevice));
    
    printDTensor(dTensorDGrad, dWidth, aHeight, "Output Gradient");
    hipblasLtMatmulDesc_t opInputGradDesc;
    hipblasLtEpilogue_t drelu = hipblasLtEpilogue_t(8 | 128);// CUBLASLT_EPILOGUE_DRELU;
    hipblasOperation_t trans = HIPBLAS_OP_T;
    size_t ww = 8;
    checkCublasStatus(hipblasLtMatmulDescCreate(&opInputGradDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opInputGradDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &trans, sizeof(trans)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opInputGradDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &drelu, sizeof(drelu)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opInputGradDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &idk, sizeof(idk)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opInputGradDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &size, sizeof(size)));
    // checkCublasStatus(cublasLtMatmulDescSetAttribute(opInputGradDesc, CUBLASLT_MATMUL_DESC_EPILOGUE_AUX_BATCH_STRIDE, &ww, sizeof(ww)));
    
    
    printf("idk: %p\n", idk);
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, opInputGradDesc, descA, descD, descB, descB, preference, 1, &heuristicResult, &returnedResults));
    printf("idk: %p\n", idk);
    algo = heuristicResult.algo;
    
    checkCublasStatus(hipblasLtMatmul(
        ltHandle, opInputGradDesc,
        &alpha,
        dTensorA, descA,
        dTensorDGrad, descD,
        &beta,
        dTensorBGrad, descB,
        dTensorBGrad, descB,
        &algo, NULL, 0, 0));
    
    
    
    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(descA));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(descB));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(descD));
    checkCublasStatus(hipblasLtMatmulDescDestroy(opDesc));
    checkCublasStatus(hipblasLtDestroy(ltHandle));
    
    return 0;
}