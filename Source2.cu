
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <sys/time.h>
#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

void initializeSeeds(uint32_t *seed1, uint32_t *seed2) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    *seed1 = tv.tv_sec;
    *seed2 = tv.tv_usec;
    for (uint8_t i = 8; i--;) {
        *seed2 *= 0xbf324c81;
        *seed1 ^= *seed2;
        *seed1 *= 0x9c7493ad;
        *seed2 ^= *seed1;
    }
}

__global__ void fillDTensor(float *dTensor, uint32_t size, uint32_t seed1, uint32_t seed2) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;
	seed1 ^= idx;
    seed1 *= 0x4ba1bb47;
    seed1 ^= seed2;
    seed1 *= 0xb7ebcb79;
    dTensor[idx] = (int32_t)seed1 * 0.0000000004656612875245797f;
}

void fillDTensor(float *dTensor, uint32_t size, uint32_t *seed1, uint32_t *seed2) {
    *seed2 *= 0xbf324c81;
    *seed1 ^= *seed2;
    *seed1 *= 0x9c7493ad;
    *seed2 ^= *seed1;
    fillDTensor<<<(size >> 10) + (size & 0x3ff), 0x400>>>(dTensor, size, *seed1, *seed2);
}

void printDTensor(float *dTensor, uint32_t width, uint32_t height, const char *label) {
    float *tensor = (float *)malloc(width * height * sizeof(float));
    checkCudaStatus(hipMemcpy(tensor, dTensor, width * height * sizeof(float), hipMemcpyDeviceToHost));
    printf("%s:\n", label);
    for (uint32_t i = 0; i < height; i++) {
        for (uint32_t j = 0; j < width; j++) {
            printf("%f ", tensor[i * width + j]);
        }
        printf("\n");
    }
    printf("\n");
    free(tensor);
}

int main() {
    uint32_t seed1, seed2;
    initializeSeeds(&seed1, &seed2);
    
    uint16_t aWidth = 4, aHeight = 3, dWidth = 2;
    
    float *dTensorA, *dTensorB, *dTensorC, *dTensorD;
    checkCudaStatus(hipMalloc((void **)&dTensorA, dWidth * aWidth * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorB, aWidth * aHeight * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorC, dWidth * aHeight * sizeof(float)));
    checkCudaStatus(hipMalloc((void **)&dTensorD, dWidth * aHeight * sizeof(float)));
    
    fillDTensor(dTensorA, dWidth * aWidth, &seed1, &seed2);
    fillDTensor(dTensorB, aWidth * aHeight, &seed1, &seed2);
    fillDTensor(dTensorC, dWidth * aHeight, &seed1, &seed2);
    
    printDTensor(dTensorA, dWidth, aWidth, "Weight");
    printDTensor(dTensorB, aWidth, aHeight, "Input");
    printDTensor(dTensorC, dWidth, aHeight, "Bias");
    
    hipblasLtHandle_t ltHandle;
    
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_BIAS;
    
    hipblasLtMatmulDesc_t opDesc;
    hipblasLtMatrixLayout_t descA, descB, descC, descD;
    
    int returnedResults;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    hipblasLtMatmulPreference_t preference;
    hipblasLtMatmulAlgo_t algo;
    
    checkCublasStatus(hipblasLtCreate(&ltHandle));
    
    checkCublasStatus(hipblasLtMatmulDescCreate(&opDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(opDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
    // checkCublasStatus(cublasLtMatmulDescSetAttribute(opDesc, CUBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue)));
    // checkCublasStatus(cublasLtMatmulDescSetAttribute(opDesc, CUBLASLT_MATMUL_DESC_BIAS_POINTER, &dTensorC, sizeof(dTensorC)));
    
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&descA, HIP_R_32F, dWidth, aWidth, dWidth));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&descB, HIP_R_32F, aWidth, aHeight, aWidth));
    // checkCublasStatus(cublasLtMatrixLayoutCreate(&descC, CUDA_R_32F, dWidth, 1, 0));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&descD, HIP_R_32F, dWidth, aHeight, dWidth));
    
    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, opDesc, descA, descB, descD, descD, preference, 1, &heuristicResult, &returnedResults));
    
    algo = heuristicResult.algo;
    float alpha = 1.0f, beta = 0.0f;
    
    checkCublasStatus(hipblasLtMatmul(
        ltHandle, opDesc,
        &alpha,
        dTensorA, descA,
        dTensorB, descB,
        &beta,
        dTensorD, descD,
        dTensorD, descD,
        &algo, NULL, 0, 0));
        
    printDTensor(dTensorD, dWidth, aHeight, "Output");
    
    checkCublasStatus(hipblasLtMatmulPreferenceDestroy(preference));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(descA));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(descB));
    // checkCublasStatus(cublasLtMatrixLayoutDestroy(descC));
    checkCublasStatus(hipblasLtMatrixLayoutDestroy(descD));
    checkCublasStatus(hipblasLtMatmulDescDestroy(opDesc));
    checkCublasStatus(hipblasLtDestroy(ltHandle));
    
    return 0;
}